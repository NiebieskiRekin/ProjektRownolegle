#include "hip/hip_runtime.h"
#include "md5_cuda.cuh"

// Constants
__device__ const uint32_t s_dev[64] = {
    7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
    5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20,
    4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
    6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21
};

__device__ const uint32_t K_dev[64] = {
    3614090360, 3905402710, 606105819, 3250441966, 4118548399, 1200080426,
    2821735955, 4249261313, 1770035416, 2336552879, 4294925233, 2304563134,
    1804603682, 4254626195, 2792965006, 1236535329, 4129170786, 3225465664,
    643717713, 3921069994, 3593408605, 38016083, 3634488961, 3889429448,
    568446438, 3275163606, 4107603335, 1163531501, 2850285829, 4243563512,
    1735328473, 2368359562, 4294588738, 2272392833, 1839030562, 4259657740,
    2763975236, 1272893353, 4139469664, 3200236656, 681279174, 3936430074,
    3572445317, 76029189, 3654602809, 3873151461, 530742520, 3299628645,
    4096336452, 1126891415, 2878612391, 4237533241, 1700485571, 2399980690,
    4293915773, 2240044497, 1873313359, 4264355552, 2734768916, 1309151649,
    4149444226, 3174756917, 718787259, 3951481745
};
__device__ const uint32_t initial_128_bit_state_dev[4] = {0x67452301, 0xefcdab89, 0x98badcfe, 0x10325476};


// Device function for left rotation
__device__ inline uint32_t leftRotate32bits_device(uint32_t n, uint32_t rotate) {
    return (n << rotate) | (n >> (32 - rotate));
}

// CUDA kernel to process multiple chunks in parallel
__global__ void process_chunks_kernel(
    const uint8_t* d_padded_message,
    uint64_t num_chunks,
    uint32_t* d_states
) {
    uint64_t chunk_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (chunk_id < num_chunks) {
        uint64_t chunk_start = chunk_id * 64;
        uint32_t blocks[16]{};
        uint32_t a0 = initial_128_bit_state_dev[0];
        uint32_t b0 = initial_128_bit_state_dev[1];
        uint32_t c0 = initial_128_bit_state_dev[2];
        uint32_t d0 = initial_128_bit_state_dev[3];
        uint32_t A = a0;
        uint32_t B = b0;
        uint32_t C = c0;
        uint32_t D = d0;

        // First, build the 16 32-bits blocks from the chunk
        for (uint8_t bid = 0; bid < 16; bid++) {
            blocks[bid] = 0;
            for (uint8_t cid = 0; cid < 4; cid++) {
                blocks[bid] = (blocks[bid] << 8) + d_padded_message[chunk_start + bid * 4 + cid];
            }
        }

        // Main "hashing" loop
        for (uint8_t i = 0; i < 64; i++) {
            uint32_t F = 0, g = 0;
            if (i < 16) {
                F = (B & C) | ((~B) & D);
                g = i;
            } else if (i < 32) {
                F = (D & B) | ((~D) & C);
                g = (5 * i + 1) % 16;
            } else if (i < 48) {
                F = B ^ C ^ D;
                g = (3 * i + 5) % 16;
            } else {
                F = C ^ (B | (~D));
                g = (7 * i) % 16;
            }

            // Update the accumulators
            F += A + K_dev[i] + blocks[g];

            A = D;
            D = C;
            C = B;
            B += leftRotate32bits_device(F, s_dev[i]);
        }
        // Store the intermediate state for this chunk
        d_states[chunk_id * 4 + 0] = A;
        d_states[chunk_id * 4 + 1] = B;
        d_states[chunk_id * 4 + 2] = C;
        d_states[chunk_id * 4 + 3] = D;
    }
}

std::array<uint8_t, 16> hash_cuda(const void* input_bs, uint64_t input_size, int threadsPerBlock) {
    const uint8_t* input = static_cast<const uint8_t*>(input_bs);

    // Preprocess the input on the host
    std::vector<uint8_t> padded_message = preprocess(input, input_size);
    uint64_t padded_size = padded_message.size();
    uint64_t num_chunks = padded_size / 64;

    // Allocate device memory for the padded message
    uint8_t* d_padded_message;
    hipMalloc(&d_padded_message, padded_size);
    hipMemcpy(d_padded_message, padded_message.data(), padded_size, hipMemcpyHostToDevice);

    // Allocate device memory for the intermediate states
    uint32_t* d_states;
    hipMalloc(&d_states, num_chunks * 4 * sizeof(uint32_t));

    // Configure the grid and block dimensions
    int numBlocks = (num_chunks + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    process_chunks_kernel<<<numBlocks, threadsPerBlock>>>(
        d_padded_message,
        num_chunks,
        d_states
    );

    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        hipFree(d_padded_message);
        hipFree(d_states);
        return std::array<uint8_t, 16>();
    }

    // Copy the intermediate states back to the host
    std::vector<uint32_t> h_states(num_chunks * 4);
    hipMemcpy(h_states.data(), d_states, num_chunks * 4 * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Perform the final sequential accumulation on the host
    std::array<uint32_t, 4> final_state = initial_128_bit_state;
    for (uint64_t i = 0; i < num_chunks; ++i) {
        final_state[0] += h_states[i * 4 + 0];
        final_state[1] += h_states[i * 4 + 1];
        final_state[2] += h_states[i * 4 + 2];
        final_state[3] += h_states[i * 4 + 3];
    }

    // Build the signature on the host
    auto signature = build_signature(final_state[0], final_state[1], final_state[2], final_state[3]);

    // Free device memory
    hipFree(d_padded_message);
    hipFree(d_states);

    return signature;
}



std::array<uint8_t, 16> hash_cuda(const void* input_bs, uint64_t input_size) {
    return hash_cuda(input_bs,input_size,1024);
}

std::array<uint8_t, 16> hash_cuda(const std::string& message) {
    return hash_cuda(message.data(), message.size());
}